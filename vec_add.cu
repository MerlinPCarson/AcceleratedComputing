#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

struct CudaBlockInfo{
  int threadsPerBlock;
  int blocksPerGrid;
};

__global__
void vecAddKernel(float * a, float * b, float * c, int size)
{
  int i = blockDim.x*blockIdx.x+threadIdx.x;

  if (i<size) c[i] = a[i] + b[i];

}

void vecAddGPU(float * h_a, float * h_b, float * h_c, int len, CudaBlockInfo * blockInfo)
{
  int size = len * sizeof(float);

  float * d_a, * d_b, * d_c;

  hipMalloc((void**)&d_a, size);
  hipMemcpy(d_a, h_a, size,  hipMemcpyHostToDevice);
  hipMalloc((void**)&d_b, size);
  hipMemcpy(d_b, h_b, size,  hipMemcpyHostToDevice);
  hipMalloc((void**)&d_c, size);

  vecAddKernel<<<blockInfo->blocksPerGrid, blockInfo->threadsPerBlock>>>(d_a, d_b, d_c, len);

  hipMemcpy(h_c, d_c, size,  hipMemcpyDeviceToHost);

  hipFree(d_a); hipFree(d_b); hipFree(d_c);
}

void vecAddCPU(float * h_a, float * h_b, float * h_c, int size)
{
  int i;
  for(i=0;i<size;++i){
    h_c[i]=h_a[i]+h_b[i];
  }

}

void fillVecs(float * h_a, float * h_b, int size)
{
  int i;
  for(i=0;i<size;++i){
    h_a[i]=i;
    h_b[i]=i;
  }

}

void printVecs(float * h_a, float * h_b, float * h_c, int size)
{
  int i;
  for(i=0;i<size;++i){
    printf("a:%f b:%f, c:%f\n",h_a[i], h_b[i], h_c[i]);
  }

}

int cudaDeviceProperties()
{
  // get number of cude devices
  int nDevices;
  hipError_t err = hipGetDeviceCount(&nDevices);
  if(err != hipSuccess){
    printf("%s\n", hipGetErrorString(err));
    return 0;
  }

  // if no cuda devices found, return error code
  if (nDevices < 1){
    return 0;
  }

  // print stats for each cuda device found
  for (int i = 0; i < nDevices; ++i){
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, i);
    printf("Device Number: %d\n", i);
    printf("  Device name: %s\n", prop.name);
    printf("  Compute Capability: %d.%d\n", prop.major, prop.minor);
    printf("  Max threads per block: %d\n", prop.maxThreadsPerBlock);
    printf("  Memory Clock Rate (KHz): %d\n", prop.memoryClockRate);
    printf("  Memory Bus Width (bits): %d\n", prop.memoryBusWidth);
    printf("  Peak Memory Bandwith (GB/s): %f\n\n",
           2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
  }

  return 1;
}

int getVectorLength(){
  int len;

  do {
    printf("Enter length of vectors to be multiplied: ");
    scanf("%d",&len);
  } while(len < 1); 

  return len;
}

void getBlockInfo(CudaBlockInfo * blockInfo, int len){

  int checkVal;

  do {
    printf("Enter number of threads per block: ");
    scanf("%d", &(blockInfo->threadsPerBlock));
    printf("Enter number of blocks per grid: ");
    scanf("%d", &(blockInfo->blocksPerGrid));
    checkVal = blockInfo->threadsPerBlock * blockInfo->blocksPerGrid;
		if (checkVal != len){
      printf("Error, try again");
    }
  } while(checkVal != len);

}

int validateBlockInfoForDevice(CudaBlockInfo  * blockInfo, int deviceNum){
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, deviceNum);

    if(prop.maxThreadsPerBlock < blockInfo->threadsPerBlock){
      printf("\nDevice %s is unable to process request!\n", prop.name);
      printf("  Max threads per block: %d\n", prop.maxThreadsPerBlock);
      printf("  Requested threads per block: %d\n", blockInfo->threadsPerBlock);
      return 0;
    }

    return 1;
}


int main(void)
{
  // identify cuda devices
  if(!cudaDeviceProperties()){
    return 1;
  }

  //int len = 1024;
  int len = getVectorLength();

  CudaBlockInfo * blockInfo = (CudaBlockInfo *)malloc(sizeof(CudaBlockInfo));
  getBlockInfo(blockInfo, len);
  //printf("%d, %d",blockInfo->threadsPerBlock, blockInfo->blocksPerGrid);

  if(!validateBlockInfoForDevice(blockInfo, 0)){
    return 1;
  }

  float * h_a = (float*)malloc(len * sizeof(float));
  float * h_b = (float*)malloc(len * sizeof(float));
  float * h_c = (float*)malloc(len * sizeof(float));

  fillVecs(h_a, h_b, len);
//  printVecs(h_a, h_b, h_c, n);

  printf("Vector addition with %d elements\n", len);

//  vecAddCPU(h_a, h_b, h_c, len);
//  printVecs(h_a, h_b, h_c, len);

  vecAddGPU(h_a, h_b, h_c, len, blockInfo);
  printVecs(h_a, h_b, h_c, len);

  return 0;
}
