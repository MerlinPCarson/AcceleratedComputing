#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <ctype.h>
#include <hip/hip_runtime.h>

#define GPUNUM (0)

struct CudaBlockInfo{
  int threadsPerBlock;
  int blocksPerGrid;
};

// Function for checking for cuda errors
void cudaCheckError(hipError_t err) {
  if(err!=hipSuccess) {
    printf("Cuda failure %s in %s line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
    exit(0);
 }
}

__global__
void vecMultKernel(float * a, float * b, float * c, int size)
{
  int i = blockDim.x*blockIdx.x+threadIdx.x;

  if (i<size) c[i] = a[i] * b[i];

}

float vecMultGPU(float * h_a, float * h_b, float * h_c, int len, CudaBlockInfo * blockInfo){

	hipEvent_t start, stop;
	float elapsedTime;

  int size = len * sizeof(float);
  float * d_a, * d_b, * d_c;
  int result = 0;

  // alocate memory and move vectors to GPU
  cudaCheckError(hipMalloc((void**)&d_a, size));
  cudaCheckError(hipMemcpy(d_a, h_a, size,  hipMemcpyHostToDevice));
  cudaCheckError(hipMalloc((void**)&d_b, size));
  cudaCheckError(hipMemcpy(d_b, h_b, size,  hipMemcpyHostToDevice));
  cudaCheckError(hipMalloc((void**)&d_c, size));

  // start timings
	hipEventCreate(&start);    
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

  // peform operation on GPU
  vecMultKernel<<<blockInfo->blocksPerGrid, blockInfo->threadsPerBlock>>>(d_a, d_b, d_c, len);
  // copy results back to CPU
  cudaCheckError(hipMemcpy(h_c, d_c, size,  hipMemcpyDeviceToHost));

  for(int i=0; i<len; ++i){
    result += h_c[i];
  }

  // end timings
	hipEventRecord(stop, 0);     
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);

  // print process time
	printf("Time to calculate results on GPU: %f ms.\n", elapsedTime);

  // free GPU memory
  hipFree(d_a); hipFree(d_b); hipFree(d_c);

	hipEventDestroy(start);
	hipEventDestroy(stop);

  return result;
}

float vecMultCPU(float * a, float * b, float * c, int len){
	hipEvent_t start, stop;
	float elapsedTime;
  int result = 0;

	hipEventCreate(&start);    
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

  for(int i=0; i<len; ++i){
    c[i] = a[i] * b[i];
    result += c[i];
  }

	hipEventRecord(stop, 0);     
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);

  // print process time
	printf("Time to calculate results on CPU: %f ms.\n", elapsedTime);

	hipEventDestroy(start);
	hipEventDestroy(stop);

  return result;
}

void fillVecs(float * a, float * b, int size){
  for(int i=0; i<size; ++i){
    a[i] = i;
    b[i] = i;
  }
}

void printVecs(float * a, float * b, float * c, int len){
  for(int i=0; i<len; ++i){
    printf("a:%f b:%f, c:%f\n",a[i], b[i], c[i]);
  }

}

int cudaDeviceProperties(){
  // get number of cude devices
  int nDevices;
  hipError_t err = hipGetDeviceCount(&nDevices);
  if(err != hipSuccess){
    printf("%s\n", hipGetErrorString(err));
    return 0;
  }

  // if no cuda devices found, return error code
  if (nDevices < 1){
    return 0;
  }

  float bytesInGiB = 1 << 30;

  // print stats for each cuda device found
  for (int i = 0; i < nDevices; ++i){
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, i);
    printf("Device Number: %d\n", i);
    printf("  Device Name: %s\n", prop.name);
    printf("  Compute Capability: %d.%d\n", prop.major, prop.minor);
    printf("  Total Global Memory (GiB): %lf\n", prop.totalGlobalMem/bytesInGiB);
    printf("  Max Threads per Block: %d\n", prop.maxThreadsPerBlock);
    printf("  Maximum x Dimension of Grid: %d\n", prop.maxGridSize[0]);
    printf("  Memory Clock Rate (KHz): %d\n", prop.memoryClockRate);
    printf("  Memory Bus Width (bits): %d\n", prop.memoryBusWidth);
    printf("  Peak Memory Bandwith (GB/s): %f\n\n",
           2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
  }

  return 1;
}

int validateBlockInfoForDevice(CudaBlockInfo  * blockInfo, int vecLen, int deviceNum){
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, deviceNum);

    if((blockInfo->threadsPerBlock*blockInfo->blocksPerGrid) < vecLen){
      printf("Number of threads per block x Number of blocks per grid < Vector Length\n");
    }
    else if(prop.maxThreadsPerBlock < blockInfo->threadsPerBlock){
      printf("\nDevice %s is unable to process request!\n", prop.name);
      printf("  Max threads per block: %d\n", prop.maxThreadsPerBlock);
      printf("  Requested threads per block: %d\n", blockInfo->threadsPerBlock);
    }
    else if(prop.maxGridSize[0] < blockInfo->blocksPerGrid){
      printf("\nDevice %s is unable to process request!\n", prop.name);
      printf("  Max blocks per grid: %d\n", prop.maxGridSize[0]);
      printf("  Requested blocks per grid: %d\n", blockInfo->blocksPerGrid);
    }
    else if(prop.totalGlobalMem < 3*(vecLen*sizeof(float))){
      printf("\nDevice %s is unable to process request!\n", prop.name);
      printf("  Total global memory is: %lu\n", prop.totalGlobalMem);
      printf("  Bytes needed for vectors: %lu\n", 3*(vecLen*sizeof(float)));
    }
    else{
      return 1;
    }

    // validation failed
    return 0;
}

void printUsage(){
  printf("\nUsage -- \n");
  printf("  VectorMultiply <length of vectors> <number of threads per block> <number of blocks per grid>\n");
}

bool isNumeric(char * str){
  int len = strlen(str);
  for(int i=0; i<len; ++i){
    if(!isdigit(str[i]))
        return false;
  }
  return true;
}

int loadArguments(int argc, char * argv[], CudaBlockInfo * blockInfo, int * vecLength){

  if(argc != 4){
    printf("\nIncorrect number of arguments!\n\n");
  }
  else if (!isNumeric(argv[1]) || !isNumeric(argv[2]) || !isNumeric(argv[3])){
    printf("\nNon-numeric value found in command line arguments\n\n");
  }
  else{
    // check if all arguments are integer values
    *vecLength = atoi(argv[1]);
    blockInfo->threadsPerBlock = atoi(argv[2]);
    blockInfo->blocksPerGrid = atoi(argv[3]);

    // validate block and thread values from arguments
    if(validateBlockInfoForDevice(blockInfo, *vecLength, GPUNUM)){
      return 1;
    }
  }

  printUsage();
  return 0;
}

int main(int argc, char * argv[])
{
  // variables for command line arguments
  int * vecLength = (int *)malloc(sizeof(int));
  CudaBlockInfo * blockInfo = (CudaBlockInfo *)malloc(sizeof(CudaBlockInfo));

  // check number of and types of command line arguments
  if(!loadArguments(argc, argv, blockInfo, vecLength)){
    return 1;
  }

  // identify cuda devices
  if(!cudaDeviceProperties()){
    return 1;
  }

  // vector variables
  float * a = (float*)malloc(*vecLength * sizeof(float));
  float * b = (float*)malloc(*vecLength * sizeof(float));
  float * c = (float*)malloc(*vecLength * sizeof(float));
  float result = 0;

  // fill vectors a and b with values
  fillVecs(a, b, *vecLength);

  printf("\nVector multiplication using CPU with %d elements:\n", *vecLength);

  result = vecMultCPU(a, b, c, *vecLength);
  printf("Result of vector multiplication on the CPU: %.2f\n", result);
  //printVecs(a, b, c, *vecLength);

  printf("\nVector multiplication using GPU with %d elements, %d threads per block and %d blocks per grid:\n", 
         *vecLength, blockInfo->threadsPerBlock, blockInfo->blocksPerGrid);

  result = vecMultGPU(a, b, c, *vecLength, blockInfo);
  printf("Result of vector multiplication on the GPU: %.2f\n", result);
  //printVecs(a, b, c, *vecLength);

  // free memory
  free(a); free(b); free(c);

  return 0;
}
