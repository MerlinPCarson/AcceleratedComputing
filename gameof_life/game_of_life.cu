/* Compile with `gcc life.c`.
 * When CUDA-fied, compile with `nvcc life.cu`
 */

#include <hip/hip_runtime.h>
#include <stdlib.h> // for rand
#include <string.h> // for memcpy
#include <stdio.h> // for printf
#include <time.h> // for nanosleep
#include <device_types.h>

#define WIDTH 60
#define HEIGHT 40
#define BLOCKSIZE 16
#define TILESIZE 16


struct CudaBlockInfo{
  dim3 threadsPerBlock;
  dim3 threadsPerTile;
  dim3 blocksPerGrid;
  dim3 tilesPerGrid;
};

// Function for checking for cuda errors
void cudaCheckError(hipError_t err) {
  if(err!=hipSuccess) {
    printf("Cuda failure %s in %s line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
    exit(EXIT_FAILURE);
 }
}

const int offsets[8][2] = {{-1, 1},{0, 1},{1, 1},
    {-1, 0},       {1, 0},
    {-1,-1},{0,-1},{1,-1}};

void fill_board(int *board, int width, int height) {
    int i;
    for (i=0; i<width*height; i++)
        board[i] = rand() % 2;
}

void print_board(int *board, int width, int height) {
    int x, y;
    for (y=0; y<height; y++) {
        for (x=0; x<width; x++) {
            char c = board[y * width + x] ? '#':' ';
            printf("%c", c);
        }
        printf("\n");
    }
    printf("-----\n");
}


void print_boards(int *boardCPU, int *boardGPU, int width, int height) {
    int x, y;
    for (y=0; y<height; ++y) {
        for (x=0; x<2*width; ++x) {
            if(x<width){
              char c = boardCPU[y * width + x] ? '#':' ';
              printf("%c", c);
            }
            else{
              if(x==width){
                printf("\t");
              }
              char c = boardGPU[y%width * width + x%width] ? '#':' ';
              printf("%c", c);
            }
        }
        printf("\n");
    }
    for(y=0; y<2; ++y){
      for(x=0; x<width+3; ++x){
        if (y==1 && x==0){
          printf("\t");
        }
        if((x<((width/2)-2)) || (x>((width/2)+2))){
          printf("-");
        }
        else if(x==(width/2)){
          if(y==0){
            printf("CPU");
          }
          else{
            printf("GPU");
          }

        }
      }
    }
    printf("\n");
}

__global__
void stepTileKernel(int *current, int *next, int width, int height){

  int offsets[8][2] = {{-1, 1},{0, 1},{1, 1},
                             {-1, 0},       {1, 0},
                             {-1,-1},{0,-1},{1,-1}};

  __shared__ int c_ds[TILESIZE+2][TILESIZE+2];

  int bx = blockIdx.x; int by = blockIdx.y;
  int tx = threadIdx.x; int ty = threadIdx.y;

  int row = by * TILESIZE + ty;
  int col = bx * TILESIZE + tx;

  //printf("(%d, %d), (%d, %d) \n", col, row, bx, by);
  //printf("(%d, %d) \n", tx, ty);

  int nx, ny;
  int num_neighbors = 0;

  // load cells data
  if((row < height) && (col < width)){
    c_ds[ty+1][tx+1] = current[row * width + col];
  }

//  tile boundry 
  if (ty == 0){
    if (tx == 0){
      nx = (col - 1 + width) % width;
      ny = (row - 1 + height) % height;
//      printf("Filling (%d,%d) with (%d, %d)\n",row, col, ny,nx);
      c_ds[ty][tx] = current[ny * width + nx];
    }
    else if(tx == (TILESIZE-1)){
      nx = (col + 1 + width) % width;
      ny = (row - 1 + height) % height;
//      printf("Filling (%d,%d) with (%d, %d)\n",row, col+2, ny,nx);
      c_ds[ty][tx+2] = current[ny * width + nx];
    }
    ny = (row - 1 + height) % height;
    c_ds[ty][tx+1] = current[ny * width + col];
  }
  else if(ty == (TILESIZE-1)){
    if (tx == 0){
      nx = (col - 1 + width) % width;
      ny = (row + 1 + height) % height;
//      printf("Filling (%d,%d) with (%d, %d)\n",row+2, col, ny,nx);
      c_ds[ty+2][tx] = current[ny * width + nx];
    }
    else if(tx == (TILESIZE-1)){
      nx = (col + 1 + width) % width;
      ny = (row + 1 + height) % height;
//      printf("Filling (%d,%d) with (%d, %d)\n",row+2, col+2, ny,nx);
      c_ds[ty+2][tx+2] = current[ny * width + nx];
    }
    ny = (row + 1 + height) % height;
    c_ds[ty+2][tx+1] = current[ny * width + col];
  }
  if (tx == 0){
    nx = (col - 1 + width) % width;
//      printf("Filling (%d,%d) with (%d, %d)\n",row+1, col, row,nx);
    c_ds[ty+1][tx] = current[row * width + nx];
  }
  else if(tx == (TILESIZE-1)){
    nx = (col + 1 + width) % width;
//    printf("Filling (%d,%d) with (%d, %d)\n",row+1, col+2, row,nx);
    c_ds[ty+1][tx+2] = current[row * width + nx];
  }

  __syncthreads();

  if ((row < height) && (col < width)){
      // count this cell's alive neighbors
      for (int i=0; i<8; i++) {
          if (c_ds[ty+1+offsets[i][0]][tx+1+offsets[i][1]]) {
              num_neighbors++;
          }
        }

      // apply the Game of Life rules to this cell
      next[row * width + col] = 0;
      if ((current[row * width + col] && num_neighbors==2) ||
          num_neighbors==3) {
        next[row * width + col] = 1;
      }

  }
//  if(tx == 0 && ty == 0){
//    for (int y=0; y<TILESIZE+2; y++) {
//        for (int x=0; x<TILESIZE+2; x++) {
//          if(x==0 || x == TILESIZE+1 || y==0 || y==TILESIZE+1){
//            printf(" ");
//          }
//          else{
//            char c = c_ds[y][x] ? '#':' ';
//            printf("%c", c);
//          }
//        }
//        printf("\n");
//    }
//    printf("-----\n");
//  }
//  if(tx == 0 && ty == 0){
//    for (int y=0; y<TILESIZE+2; y++) {
//        for (int x=0; x<TILESIZE+2; x++) {
//            char c = c_ds[y][x] ? '#':' ';
//            printf("%c", c);
//        }
//        printf("\n");
//    }
//    printf("-----\n");
//  }

  __syncthreads();

}

__global__
void stepKernel(int *current, int *next, int width, int height){

  int offsets[8][2] = {{-1, 1},{0, 1},{1, 1},
                             {-1, 0},       {1, 0},
                             {-1,-1},{0,-1},{1,-1}};

  int row = blockDim.y*blockIdx.y+threadIdx.y;
  int col = blockDim.x*blockIdx.x+threadIdx.x;

//    printf("(%d, %d) \n", col, row);

  int nx, ny;
  int num_neighbors = 0;

  if ((row < height) && (col < width)){
      // count this cell's alive neighbors
      for (int i=0; i<8; i++) {
          // To make the board torroidal, we use modular arithmetic to
          // wrap neighbor coordinates around to the other side of the
          // board if they fall off.
          nx = (col + offsets[i][0] + width) % width;
          ny = (row + offsets[i][1] + height) % height;
          if (current[ny * width + nx]) {
              num_neighbors++;
          }
        }

      // apply the Game of Life rules to this cell
      next[row * width + col] = 0;
      if ((current[row * width + col] && num_neighbors==2) ||
          num_neighbors==3) {
          next[row * width + col] = 1;
      }

  }
}

float stepGPU(int *h_current, int *h_next, int width, int height, CudaBlockInfo * blockInfo ) {
  // timing vars
	hipEvent_t start, stop;
	float elapsedTime;

  size_t size = width * height * sizeof(int);
  int *d_current, *d_next;

  // alocate memory and move vectors to GPU
  cudaCheckError(hipMalloc((void**)&d_current, size));
  cudaCheckError(hipMemcpy(d_current, h_current, size,  hipMemcpyHostToDevice));
  cudaCheckError(hipMalloc((void**)&d_next, size));

  // start timings
	hipEventCreate(&start);    
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

  // peform operation on GPU
  printf("%d,%d,%d:%d,%d,%d\n", blockInfo->threadsPerBlock.x, blockInfo->threadsPerBlock.y, blockInfo->threadsPerBlock.z, blockInfo->blocksPerGrid.x, blockInfo->blocksPerGrid.y, blockInfo->blocksPerGrid.z);
  //stepKernel<<<blockInfo->blocksPerGrid, blockInfo->threadsPerBlock>>>(d_current, d_next, width, height);
  stepTileKernel<<<blockInfo->tilesPerGrid, blockInfo->threadsPerTile>>>(d_current, d_next, width, height);
  // copy results back to CPU
  cudaCheckError(hipMemcpy(h_next, d_next, size,  hipMemcpyDeviceToHost));

  // end timings
	hipEventRecord(stop, 0);     
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);

  // free GPU memory
  hipFree(d_current); hipFree(d_next);

	hipEventDestroy(start);
	hipEventDestroy(stop);

  
  return elapsedTime;
}

float step(int *current, int *next, int width, int height) {
    // timing vars
    hipEvent_t start, stop;
    float elapsedTime;
    // coordinates of the cell we're currently evaluating
    int x, y;
    // offset index, neighbor coordinates, alive neighbor count
    int i, nx, ny, num_neighbors;

    // start timings
    hipEventCreate(&start);    
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    // write the next board state
    for (y=0; y<height; y++) {
        for (x=0; x<width; x++) {

            // count this cell's alive neighbors
            num_neighbors = 0;
            for (i=0; i<8; i++) {
                // To make the board torroidal, we use modular arithmetic to
                // wrap neighbor coordinates around to the other side of the
                // board if they fall off.
                nx = (x + offsets[i][0] + width) % width;
                ny = (y + offsets[i][1] + height) % height;
                if (current[ny * width + nx]) {
                    num_neighbors++;
                }
            }

            // apply the Game of Life rules to this cell
            next[y * width + x] = 0;
            if ((current[y * width + x] && num_neighbors==2) ||
                    num_neighbors==3) {
                next[y * width + x] = 1;
            }
        }
    }

    // end timings
	  hipEventRecord(stop, 0);     
	  hipEventSynchronize(stop);
	  hipEventElapsedTime(&elapsedTime, start, stop);

    return elapsedTime;
}

int main(int argc, const char *argv[]) {


    // parse the width and height command line arguments, if provided
    int width, height, numIters, out;
    if (argc < 3) {
        printf("usage: life iterations 1=print"); 
        exit(1);
    }
    numIters = atoi(argv[1]);
    out = atoi(argv[2]);
    if (argc == 5) {
        width = atoi(argv[3]);
        height = atoi(argv[4]);
        printf("Running %d iterations at %d by %d pixels.\n", numIters, width, height);
    } else {
        width = WIDTH;
        height = HEIGHT;
    }

    // GPU vars
    //bool on_gpu = true;
    CudaBlockInfo * blockInfo = (CudaBlockInfo *)malloc(sizeof(CudaBlockInfo));
    dim3 blocksPerGrid(ceil((float)width/BLOCKSIZE), ceil((float)height/BLOCKSIZE), 1);
    dim3 tilesPerGrid(ceil((float)width/TILESIZE), ceil((float)height/TILESIZE), 1);
    dim3 threadsPerBlock(BLOCKSIZE, BLOCKSIZE, 1);
    dim3 threadsPerTile(TILESIZE, TILESIZE, 1);
    blockInfo->threadsPerTile = threadsPerTile;
    blockInfo->threadsPerBlock = threadsPerBlock;
    blockInfo->blocksPerGrid = blocksPerGrid;
    blockInfo->tilesPerGrid = tilesPerGrid;
    struct timespec delay = {0, 125000000}; // 0.125 seconds
    struct timespec remaining;

    float procTime = 0;
    float totalProcTimeCPU = 0;
    float totalProcTimeGPU = 0;

    // The two boards 
    int currIter = 0;
    int *start, *current, *next;
    int *currentGPU, *nextGPU;

    size_t board_size = sizeof(int) * width * height;
    start = (int *) malloc(board_size); // same as: int current[width * height];
    current = (int *) malloc(board_size); // same as: int current[width * height];
    next = (int *) malloc(board_size);    // same as: int next[width *height];
    currentGPU = (int *) malloc(board_size); // same as: int current[width * height];
    nextGPU = (int *) malloc(board_size);    // same as: int next[width *height];
 
    printf("Initializing boards\n"); 
    fill_board(start, width, height);
    memcpy(current, start, board_size);
    memcpy(currentGPU, start, board_size);


    // Run on CPU
    while (currIter<numIters) {
        ++currIter;
        if (out==1)
            print_boards(current, currentGPU, width, height);

        //evaluate the `current` board, writing the next generation into `next`.
        procTime = step(current, next, width, height);
        totalProcTimeCPU += procTime;

        // Copy the next state, that step() just wrote into, to current state
        memcpy(current, next, board_size);

        // print process time
        //printf("Time to calculate results on CPU: %f ms.\n", procTime);

        // copy the `next` to CPU and into `current` to be ready to repeat the process
        procTime = stepGPU(currentGPU, nextGPU, width, height, blockInfo);
        totalProcTimeGPU += procTime;

        // Copy the next state, that step() just wrote into, to current state
        memcpy(currentGPU, nextGPU, board_size);

        // print process time
        //printf("Time to calculate results on GPU: %f ms.\n", procTime);

        // We sleep only because textual output is slow and the console needs
        // time to catch up. We don't sleep in the graphical X11 version.
        if (out==1)
            nanosleep(&delay, &remaining);
    }


    printf("Average processing time on CPU is %f ms.\n", (totalProcTimeCPU/numIters));
    printf("Average processing time on GPU is %f ms.\n", (totalProcTimeGPU/numIters));

    free(blockInfo);
    free(start);
    free(current);
    free(next);
    free(currentGPU);
    free(nextGPU);

    return 0;
}
