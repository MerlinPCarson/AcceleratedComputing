/* Compile with `gcc life.c`.
 * When CUDA-fied, compile with `nvcc life.cu`
 */

#include <hip/hip_runtime.h>
#include <stdlib.h> // for rand
#include <string.h> // for memcpy
#include <stdio.h> // for printf
#include <time.h> // for nanosleep
#include <device_types.h>

#define WIDTH 60
#define HEIGHT 40
#define BLOCKSIZE 16


struct CudaBlockInfo{
  dim3 threadsPerBlock;
  dim3 blocksPerGrid;
};

// Function for checking for cuda errors
void cudaCheckError(hipError_t err) {
  if(err!=hipSuccess) {
    printf("Cuda failure %s in %s line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
    exit(EXIT_FAILURE);
 }
}

const int offsets[8][2] = {{-1, 1},{0, 1},{1, 1},
    {-1, 0},       {1, 0},
    {-1,-1},{0,-1},{1,-1}};

void fill_board(int *board, int width, int height) {
    int i;
    for (i=0; i<width*height; i++)
        board[i] = rand() % 2;
}

void print_board(int *board, int width, int height) {
    int x, y;
    for (y=0; y<height; y++) {
        for (x=0; x<width; x++) {
            char c = board[y * width + x] ? '#':' ';
            printf("%c", c);
        }
        printf("\n");
    }
    printf("-----\n");
}

__global__
void stepKernel(int *current, int *next, int width, int height){

  int offsets[8][2] = {{-1, 1},{0, 1},{1, 1},
                             {-1, 0},       {1, 0},
                             {-1,-1},{0,-1},{1,-1}};

  int row = blockDim.y*blockIdx.y+threadIdx.y;
  int col = blockDim.x*blockIdx.x+threadIdx.x;

//    printf("(%d, %d) \n", col, row);

  int nx, ny;
  int num_neighbors = 0;

  if ((row < height) && (col < width)){
      // count this cell's alive neighbors
      for (int i=0; i<8; i++) {
          // To make the board torroidal, we use modular arithmetic to
          // wrap neighbor coordinates around to the other side of the
          // board if they fall off.
          nx = (col + offsets[i][0] + width) % width;
          ny = (row + offsets[i][1] + height) % height;
          if (current[ny * width + nx]) {
              num_neighbors++;
          }
        }

      // apply the Game of Life rules to this cell
      next[row * width + col] = 0;
      if ((current[row * width + col] && num_neighbors==2) ||
          num_neighbors==3) {
          next[row * width + col] = 1;
      }

  }
}

float stepGPU(int *h_current, int *h_next, int width, int height, CudaBlockInfo * blockInfo ) {
  // timing vars
	hipEvent_t start, stop;
	float elapsedTime;

  size_t size = width * height * sizeof(int);
  int *d_current, *d_next;

  // alocate memory and move vectors to GPU
  cudaCheckError(hipMalloc((void**)&d_current, size));
  cudaCheckError(hipMemcpy(d_current, h_current, size,  hipMemcpyHostToDevice));
  cudaCheckError(hipMalloc((void**)&d_next, size));

  // start timings
	hipEventCreate(&start);    
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

  // peform operation on GPU
  //printf("%d,%d,%d:%d,%d,%d\n", blockInfo->threadsPerBlock.x, blockInfo->threadsPerBlock.y, blockInfo->threadsPerBlock.z, blockInfo->blocksPerGrid.x, blockInfo->blocksPerGrid.y, blockInfo->blocksPerGrid.z);
  stepKernel<<<blockInfo->blocksPerGrid, blockInfo->threadsPerBlock>>>(d_current, d_next, width, height);
  // copy results back to CPU
  cudaCheckError(hipMemcpy(h_next, d_next, size,  hipMemcpyDeviceToHost));
//  printf("printing board");
//  print_board(h_next);
//  printf("done printing board");

  // end timings
	hipEventRecord(stop, 0);     
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);

  // free GPU memory
  hipFree(d_current); hipFree(d_next);

	hipEventDestroy(start);
	hipEventDestroy(stop);

  
  return elapsedTime;
}

float step(int *current, int *next, int width, int height) {
    // timing vars
    hipEvent_t start, stop;
    float elapsedTime;
    // coordinates of the cell we're currently evaluating
    int x, y;
    // offset index, neighbor coordinates, alive neighbor count
    int i, nx, ny, num_neighbors;

    // start timings
    hipEventCreate(&start);    
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    // write the next board state
    for (y=0; y<height; y++) {
        for (x=0; x<width; x++) {

            // count this cell's alive neighbors
            num_neighbors = 0;
            for (i=0; i<8; i++) {
                // To make the board torroidal, we use modular arithmetic to
                // wrap neighbor coordinates around to the other side of the
                // board if they fall off.
                nx = (x + offsets[i][0] + width) % width;
                ny = (y + offsets[i][1] + height) % height;
                if (current[ny * width + nx]) {
                    num_neighbors++;
                }
            }

            // apply the Game of Life rules to this cell
            next[y * width + x] = 0;
            if ((current[y * width + x] && num_neighbors==2) ||
                    num_neighbors==3) {
                next[y * width + x] = 1;
            }
        }
    }

    // end timings
	  hipEventRecord(stop, 0);     
	  hipEventSynchronize(stop);
	  hipEventElapsedTime(&elapsedTime, start, stop);

    return elapsedTime;
}

int main(int argc, const char *argv[]) {


    // parse the width and height command line arguments, if provided
    int width, height, iters, out;
    if (argc < 3) {
        printf("usage: life iterations 1=print"); 
        exit(1);
    }
    iters = atoi(argv[1]);
    out = atoi(argv[2]);
    if (argc == 5) {
        width = atoi(argv[3]);
        height = atoi(argv[4]);
        printf("Running %d iterations at %d by %d pixels.\n", iters, width, height);
    } else {
        width = WIDTH;
        height = HEIGHT;
    }

    // GPU vars
    //bool on_gpu = true;
    CudaBlockInfo * blockInfo = (CudaBlockInfo *)malloc(sizeof(CudaBlockInfo));
    dim3 threadsPerBlock(ceil((float)width/BLOCKSIZE), ceil((float)height/BLOCKSIZE), 1);
    dim3 blocksPerGrid(BLOCKSIZE, BLOCKSIZE, 1);
    blockInfo->threadsPerBlock = threadsPerBlock;
    blockInfo->blocksPerGrid = blocksPerGrid;
    struct timespec delay = {0, 125000000}; // 0.125 seconds
    struct timespec remaining;

    float procTime = 0;
    float totalProcTimeCPU = 0;
    float totalProcTimeGPU = 0;

    // The two boards 
    int *current, *next, many=0;

    size_t board_size = sizeof(int) * width * height;
    current = (int *) malloc(board_size); // same as: int current[width * height];
    next = (int *) malloc(board_size);    // same as: int next[width *height];
 
    printf("Initializing board for CPU\n"); 
    fill_board(current, width, height);


    // Run on CPU
    while (many<iters) {
        many++;
        if (out==1)
            print_board(current, width, height);

        //evaluate the `current` board, writing the next generation into `next`.
        procTime = step(current, next, width, height);
        totalProcTimeCPU += step(current, next, width, height);

        // Copy the next state, that step() just wrote into, to current state
        memcpy(current, next, board_size);

        // print process time
        //printf("Time to calculate results on CPU: %f ms.\n", procTime);

        // We sleep only because textual output is slow and the console needs
        // time to catch up. We don't sleep in the graphical X11 version.
        if (out==1)
            nanosleep(&delay, &remaining);
    }

    many = 0;

    // Initialize the global "current".
    printf("Initializing board for GPU\n"); 
    nanosleep(&delay, &remaining);
    fill_board(current, width, height);

    while (many<iters) {
        many++;
        if (out==1)
            print_board(current, width, height);

        // copy the `next` to CPU and into `current` to be ready to repeat the process
        procTime = stepGPU(current, next, width, height, blockInfo);
        totalProcTimeGPU += procTime;

        // Copy the next state, that step() just wrote into, to current state
        memcpy(current, next, board_size);

        // print process time
        //printf("Time to calculate results on GPU: %f ms.\n", procTime);

        // We sleep only because textual output is slow and the console needs
        // time to catch up. We don't sleep in the graphical X11 version.
        if (out==1)
            nanosleep(&delay, &remaining);
    }

    printf("Average processing time on CPU is %f ms.\n", (totalProcTimeCPU/iters));
    printf("Average processing time on GPU is %f ms.\n", (totalProcTimeGPU/iters));

    free(blockInfo);
    free(current);
    free(next);

    return 0;
}
